#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define THREADS_PER_BLOCK 512

//function declarations
unsigned int getmax(unsigned int *, unsigned int);
__global__ void get_max(unsigned int *num, unsigned int size);

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array

    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
    //size is number of threads total
    size = atol(argv[1]);

    //calculates number of blocks
    unsigned int NUM_BLOCKS = size/THREADS_PER_BLOCK;

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1
    for( i = 0; i < size; i++){
      numbers[i] = rand()  % size;
    }
    //create device pointers
    unsigned int *d_numbers;
    //transfer array to device memory
    hipMalloc((void**) &d_numbers, size * sizeof(unsigned int));
    hipMemcpy(d_numbers, numbers, size * sizeof(unsigned int), hipMemcpyHostToDevice);
    //sequential
    printf(" The maximum number in the array is: %u\n", getmax(numbers, size));
    //parallel
    get_max<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_numbers, size);
    hipMemcpy(numbers, d_numbers, size * sizeof(unsigned int), hipMemcpyDeviceToHost);
    for( i = 0; i < size; i++){
      if(numbers[i] > numbers[0]){
        printf("element in %d: %u\n", i, numbers[i]);
      }
    }
     printf("The max integer in the array is: %d\n", numbers[0]);
    //free device matrices
    hipFree(d_numbers);
    free(numbers);
    exit(0);
}

__global__ void get_max(unsigned int* num, unsigned int size){
  unsigned int temp;
  unsigned int index = threadIdx.x + (blockDim.x * blockIdx.x);
  unsigned int nTotalThreads = size;
  unsigned int i = 0;

  while(nTotalThreads > 1){
    unsigned int tenPoint = nTotalThreads / 10;	// divide by ten
    // only the first half of the threads will be active.

    if(index < tenPoint){
      for(i = 1; i < 10; i++){
        temp = num[index + tenPoint*i];
        //compare to "0" index
        if(temp > num[index]){
          num[index] = temp;
        }
      }
    }
    /*
    if (index < halfPoint){
      temp = num[ index + halfPoint ];
      if (temp > num[ index ]) {
        num[index] = temp;
      }
    }*/
    __syncthreads();


    nTotalThreads = (nTotalThreads / 10) * 9;	// divide by two.
  }
}

/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
unsigned int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}
